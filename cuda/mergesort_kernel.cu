#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <algorithm>
#include <iterator>
#include <chrono>

using namespace std;

/**
 * mergesort.cu
 * a one-file c++ / cuda program for performing mergesort on the GPU
 * While the program execution is fairly slow, most of its runnning time
 *  is spent allocating memory on the GPU.
 * For a more complex program that performs many calculations,
 *  running on the GPU may provide a significant boost in performance
 */

vector<int> buildArrFromFile(string fileName);


// data[], size, threads, blocks, 
void mergesort(int*, int, dim3, dim3);

// A[]. B[], size, width, slices, nThreads
__global__ void gpu_mergesort(long*, long*, long, long, long, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(long*, long*, long, long, long);



#define min(a, b) (a < b ? a : b)


bool verbose;
int main(int argc, char** argv) {

	dim3 threadsPerBlock;
	dim3 blocksPerGrid;

	threadsPerBlock.x = 32;
	threadsPerBlock.y = 1;
	threadsPerBlock.z = 1;

	blocksPerGrid.x = 24;
	blocksPerGrid.y = 1;
	blocksPerGrid.z = 1;

	auto startTime = chrono::steady_clock::now();
	auto endTime = chrono::steady_clock::now();
	std::chrono::duration<double, std::milli> durationMs;

	// Read numbers from file
	string fileName = "array_2M_range_1000.txt";
	vector<int> data = buildArrFromFile(fileName);
	int size = data.size();

	// merge-sort the data
	startTime = chrono::steady_clock::now();
	mergesort(&data[0], size, threadsPerBlock, blocksPerGrid);
	endTime = chrono::steady_clock::now();


	durationMs = endTime - startTime;
	std::cout << "The time to run this algorithm is: " << durationMs.count();

	
	for (int i = 0; i < size-1; i++) {
		if (data[i] > data[i + 1]) {
			cout << "Not Sorted" << endl;
			break;
		}
	}
}

void mergesort(int* data, int size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

	// Allocate two arrays on the GPU
	// we switch back and forth between them during the sort
	long* D_data;
	long* D_swp;
	dim3* D_threads;
	dim3* D_blocks;
	hipError_t cudaStatus;
		// Actually allocate the two arrays

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		exit(1);
	}

	int megabytesToUse = 160;

	size_t newHeapSize = 1024 * 1000 * megabytesToUse;
	cudaStatus = hipDeviceSetLimit(hipLimitMallocHeapSize, 
		newHeapSize);
	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceSetLimit failed!");
		exit(1);
	}
	//printf("Adjusted heap size to be %d\n", (int)newHeapSize);



	cudaStatus = hipMalloc((void**)&D_data, size * sizeof(long) * 2);
	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		exit(1);
	}
	cudaStatus = hipMalloc((void**)&D_swp, size * sizeof(long) * 2);
	if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "cudadevicesynchronize returned error code %d after launching addkernel!\n", cudastatus);
        exit(1);
    }

	// Copy from our input list into the first array
	cudaStatus = hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);

	//
	// Copy the thread / block info to the GPU as well
	//
	cudaStatus = hipMalloc((void**)&D_threads, sizeof(dim3));
	cudaStatus = hipMalloc((void**)&D_blocks, sizeof(dim3));

	cudaStatus = hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);


	long* A = D_data;
	long* B = D_swp;

	long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
		blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

	auto startTime = chrono::steady_clock::now();
	auto endTime = chrono::steady_clock::now();
	std::chrono::duration<double, std::milli> durationMs;
	//
	// Slice up the list and give pieces of it to each thread, letting the pieces grow
	// bigger and bigger until the whole list is sorted
	//

	startTime = chrono::steady_clock::now();
	for (int width = 2; width < (size << 1); width <<= 1) {
		long slices = size / ((nThreads)* width) + 1;

		// Actually call the kernel
		gpu_mergesort << <blocksPerGrid, threadsPerBlock >> > (A, B, size, width, slices, D_threads, D_blocks);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			exit(1);
		}

		// Switch the input / output arrays instead of copying them around
		A = A == D_data ? D_swp : D_data;
		B = B == D_data ? D_swp : D_data;
	}
	endTime = chrono::steady_clock::now();
	durationMs = endTime - startTime;
	std::cout << "The time to run the GPU is: " << durationMs.count() << endl;
	//
	// Get the list back from the GPU
 	cudaStatus = hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);

	// Free the GPU memory
	cudaStatus = hipFree(A);
	cudaStatus = hipFree(B);

	// hipDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		exit(1);
	}

}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
	int x;
	return threadIdx.x +
		threadIdx.y * (x = threads->x) +
		threadIdx.z * (x *= threads->y) +
		blockIdx.x  * (x *= threads->z) +
		blockIdx.y  * (x *= blocks->z) +
		blockIdx.z  * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
	unsigned int idx = getIdx(threads, blocks);
	long start = width * idx*slices,
		middle,
		end;

	for (long slice = 0; slice < slices; slice++) {
		if (start >= size)
			break;

		middle = min(start + (width >> 1), size);
		end = min(start + width, size);
		
		gpu_bottomUpMerge(source, dest, start, middle, end);
		start += width;
	}
}

// Finally, sort something
// gets called by gpu_mergesort() for each slice
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
	long i = start;
	long j = middle;
	for (long k = start; k < end; k++) {
		if (i < middle && (j >= end || source[i] < source[j])) {
			dest[k] = source[i];
			i++;
		}
		else {
			dest[k] = source[j];
			j++;
		}
	}
}



vector<int> buildArrFromFile(string fileName) {
	string line;
	vector<int> vect;
	ifstream aFileToSort(fileName);
	while (getline(aFileToSort, line)) {
		vect.push_back(stoi(line));
	}

	return (vect);
}

