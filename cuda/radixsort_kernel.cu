
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/


  // Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <algorithm>
#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <string>
#include <vector>
#include <chrono>

#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>
#include "test/test_util.h"

using namespace hipcub;
using namespace std;

// Caching allocator for device memory
CachingDeviceAllocator  g_allocator(true);  

vector<int> buildArrFromFile(string fileName) {
	string line;
	vector<int> vect;
	ifstream aFileToSort(fileName);
	while (getline(aFileToSort, line)) {
		vect.push_back(stoi(line));
	}
	return (vect);
}

int main(int argc, char** argv)
{
	// Initialize device
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		exit(1);
	}
	

	// Initialize the timer
	auto startTime = chrono::steady_clock::now();
	auto endTime = chrono::steady_clock::now();
	std::chrono::duration<double, std::milli> durationMs;

	// Initialize vector
	vector<int> vect;
	string fileName = "array_2M_range_1000.txt";
	vect = buildArrFromFile(fileName);
	int num_vals = vect.size();

	// Capturing the execute time
	startTime = chrono::steady_clock::now();

	// Allocate device arrays
	DoubleBuffer<int>   d_vals;
	CubDebugExit(g_allocator.DeviceAllocate((void**)&d_vals.d_buffers[0], sizeof(int) * num_vals));
	CubDebugExit(g_allocator.DeviceAllocate((void**)&d_vals.d_buffers[1], sizeof(int) * num_vals));
	
	
	// Allocate temporary storage
	size_t  temp_storage_bytes = 0;
	void    *d_temp_storage = NULL;

	CubDebugExit(DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_vals, num_vals));
	CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

	// Initialize device arrays
	CubDebugExit(hipMemcpy(d_vals.d_buffers[d_vals.selector], &vect[0], sizeof(int) * num_vals, hipMemcpyHostToDevice));

	// Run
	CubDebugExit(DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_vals, num_vals));

	// Allocate array on host
	int *h_sorted = new int[num_vals];

	// Copy data back
	hipMemcpy(h_sorted, d_vals.Current(), sizeof(int) * num_vals, hipMemcpyDeviceToHost);

	// Capturing the execute time
	endTime = chrono::steady_clock::now();

	durationMs = endTime - startTime;
	std::cout << "The time to run this algorithm is: " << durationMs.count() << endl;

	bool sorted = true;
	for (int i = 0; i < num_vals - 1; i++) {
		if (h_sorted[i] > h_sorted[i + 1]) {
			cout << "Not Sorted" << endl;
			sorted = false;
			break;
		}
	}
	if (sorted)
		cout << "The Array Is Sorted" << endl;

	printf("\n");

	// Cleanup
	if (h_sorted) delete[] h_sorted;
	if (d_vals.d_buffers[0]) CubDebugExit(g_allocator.DeviceFree(d_vals.d_buffers[0]));
	if (d_vals.d_buffers[1]) CubDebugExit(g_allocator.DeviceFree(d_vals.d_buffers[1]));
	if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
	printf("\n\n");

	return 0;
}



